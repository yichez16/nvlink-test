#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>



// warm up l2 cache 
__global__ void arrayToL2Cache(int* array, int size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size)
    {
        // Access and perform dummy operations on the array elements
        int value = array[tid];
        value *= 2;
        array[tid] = value;
    }
}

// copy data from remote to local. 
// One kernel only contains one thread.
__global__ void copyKernel_single(int* local, int* remote, int threadID)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    clock_t startClock = clock();
    if (tid == threadID)
    {
        local[tid] = remote[tid];
    }
    clock_t stopClock = clock();
    clock_t elapsedTime = stopClock - startClock;
    printf("ThreadID: %d,Elapsed Time: %llu cycles\n", threadID, elapsedTime);


}

// copy data from remote to local. 
// One kernel only contains two thread.
__global__ void copyKernel_two(int* local, int* remote, int threadID1, int threadID2)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    clock_t startClock = clock();
    if (tid == threadID1 or tid ==threadID2 )
    {
        local[tid] = remote[tid];
    }
    clock_t stopClock = clock();
    clock_t elapsedTime = stopClock - startClock;
    printf("ThreadID: %d,Elapsed Time: %llu cycles\n", threadID2, elapsedTime);

}