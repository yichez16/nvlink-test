#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>



// warm up l2 cache 
__global__ void arrayToL2Cache(int* array, int size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size)
    {
        // Access and perform dummy operations on the array elements
        int value = array[i];
        value *= 2;
        array[i] = value;
    }
}