#include <hip/hip_runtime.h>
#include <iostream>
#include "kernel.cu"
#include <stdio.h>

#define ARRAY_SIZE 1000000 // l2 cache size = 4MB with 128 Bytes cache line size



int main()
{
    int* devArrayLocal;
    int* devArrayRemote;
    const int numElements = ARRAY_SIZE * sizeof(int);
    int hostArrayLocal[ARRAY_SIZE];
    int hostArrayRemote[ARRAY_SIZE];

    // Initialize hostArrayLocal and hostArrayRemote with sequential values
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        hostArrayLocal[i] = i;
        hostArrayRemote[i] = i+1;
    }

    // Allocate memory for the arrays on the local GPU 0
    hipSetDevice(0);
    hipMalloc(&devArrayLocal, numElements);

    // Allocate memory for the array on the remote GPU 1
    hipSetDevice(1);
    hipMalloc(&devArrayRemote, numElements);

    // Copy hostArrayLocal to devArrayLocal on the local GPU 0
    hipSetDevice(0);
    hipMemcpy(devArrayLocal, hostArrayLocal, numElements, hipMemcpyHostToDevice);

    // Copy hostArrayRemote to devArrayRemote on the remote GPU 1
    hipSetDevice(1);
    hipMemcpy(devArrayRemote, hostArrayRemote, numElements, hipMemcpyHostToDevice);

    // Launch the kernel on the local GPU 0 to perform operations on the local array
    hipSetDevice(0);
    arrayToL2Cache<<<1, 32>>>(devArrayLocal, ARRAY_SIZE);

    // Launch the kernel on the remote GPU 1 to perform operations on the remote array
    hipSetDevice(1);
    arrayToL2Cache<<<1, 32>>>(devArrayRemote, ARRAY_SIZE);

    // Synchronize the local GPU 0 to ensure the kernel execution is completed
    hipSetDevice(0);
    hipDeviceSynchronize();

    // Make sure local gpu 0 can acess remote gpu 1
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(1, 0); 
    // copyKernel_single <<<1, 1>>>(devArrayLocal, devArrayRemote, 0);
    // copyKernel_single <<<1, 1>>>(devArrayLocal, devArrayRemote, 1);
    copyKernel_two <<<1, 2>>>(devArrayLocal, devArrayRemote, 0, 1);




    // Copy devArrayLocal back to hostArrayLocal on the local GPU 0
    hipMemcpy(hostArrayLocal, devArrayLocal, numElements, hipMemcpyDeviceToHost);

    // Copy devArrayRemote back to hostArrayRemote on the remote GPU 1
    hipSetDevice(1);
    hipMemcpy(hostArrayRemote, devArrayRemote, numElements, hipMemcpyDeviceToHost);




    // Print the modified values from both local and remote arrays
    std::cout << "Local Array: ";
    for (int i = 0; i < 10; i++)
    {
        std::cout << hostArrayLocal[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Remote Array: ";
    for (int i = 0; i < 10; i++)
    {
        std::cout << hostArrayRemote[i] << " ";
    }
    std::cout << std::endl;

    // Free the allocated memory
    hipFree(devArrayRemote);
    hipFree(devArrayLocal);

    return 0;
}